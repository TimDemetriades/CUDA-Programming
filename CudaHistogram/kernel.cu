#include "hip/hip_runtime.h"
// CUDA Histogram Computation
// Tim Demetriades
// CPE 810 - GPU & Multicore Programming
// Professor Feng
// Stevens Institute of Technology

#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime_api.h"

//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include <stdio.h>
#include <cstdlib>
#include <time.h>

// Statically allocate shared memory
//#define SHARED_MEM_SIZE 16 * 16	// Dimensions of each block/tile

// Kernel function for Histogram Computation
__global__ void Histogram_Computation(unsigned int* device_input, unsigned int* device_bins, unsigned int input_size, unsigned int num_bins) {

	printf("Test");

	// Get thread id
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int DIV = (input_size + num_bins - 1) / num_bins;

	//printf("tid = %d\n", tid);
	//printf("input_size = %d\n", input_size);

	// Boundary condition
	if (tid < input_size) {
		int bin = device_input[tid] / DIV;
		atomicAdd(&device_bins[bin], 1);
		printf("Test");
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char* argv[]) {
	if (argc == 4) {		// 4 arguments expected (filename, -i, <BinNum>, <VecDim>)
		printf("\n");
		printf("There will be %s bins \n", argv[2]);
		printf("Input vector is %s elements long \n", argv[3]);
	}
	else if (argc > 4) {
		printf("Too many arguments provided. \n");
		printf("Enter arguments like this: \n");
		printf("-i <BinNum> <VecDim> \n");
		exit(EXIT_FAILURE);
	}
	else {
		printf("3 arguments expected. \n");
		printf("Enter arguments like this: \n");
		printf("-i <BinNum> <VecDim> \n");
		exit(EXIT_FAILURE);
	}

	printf("\nStarting Histogram Computation on GPU\n");

	// Set number of bins
	int num_bins_string;
	num_bins_string = atoi(argv[2]);
	int num_bins = num_bins_string;
	//int num_bins = atoi(argv[2]);

	// Set number of input elements
	int input_size_string;
	input_size_string = atoi(argv[3]);
	int input_size = input_size_string;
	//int input_size = atoi(argv[3]);

	// Set number of elements per bin
	int bin_size = ceil(input_size / num_bins);		// Padded in case num of bins does not equally divide num of input elements
	//(input_size + num_bins - 1) / num_bins   ->   alternate method

	// Size in bytes of input vector
	size_t input_bytes = input_size * sizeof(int);

	// Size in bytes of bins
	size_t bin_bytes = num_bins * sizeof(int);

	// Allocate host memory for input vector and bins
	unsigned int* host_input;
	unsigned int* host_bins;
	host_input = (unsigned int*)malloc(input_bytes);
	host_bins = (unsigned int*)malloc(bin_bytes);

	// Allocate device memory for input vector and bins
	unsigned int* device_input;
	unsigned int* device_bins;
	checkCudaErrors(hipMalloc((void**)&device_input, input_bytes));
	checkCudaErrors(hipMalloc((void**)&device_bins, bin_bytes));

	// Initialize input vector with ints between 0~1024
	for (int i = 0; i < input_size; i++) {
		host_input[i] = rand() % 1025;
	}

	// Initialize bins with 0s
	for (int i = 0; i < num_bins; i++) {
		host_bins[i] = 0;
	}

	// Print input vector
	printf("\nInput vector:\n");
	for (int i = 0; i < input_size; i++) {
		printf("%d\t", host_input[i]);
	}

	// Print input vector
	printf("\nBins:\n");
	for (int i = 0; i < num_bins; i++) {
		printf("%d\t", host_bins[i]);
	}

	// Copy matrix values from host to device
	checkCudaErrors(hipMemcpy(device_input, host_input, input_bytes, hipMemcpyHostToDevice));		// dest, source, size in bytes, direction of transfer
	checkCudaErrors(hipMemcpy(device_bins, host_bins, bin_bytes, hipMemcpyHostToDevice));		// dest, source, size in bytes, direction of transfer

	//// Print input vector
	//printf("\nInput vector - Device:\n");
	//for (int i = 0; i < input_size; i++) {
	//	printf("%d\t", device_bins[i]);
	//}

	// Set Grid and Block sizes
	int block_size = 512;		// Threads per block
	int grid_size = ceil(input_size / block_size) + 1;
	dim3 dim_block(block_size, block_size);
	dim3 dim_grid(grid_size, grid_size);

	// Record the start event (for timing GPU calculations)
	hipStream_t stream;
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	checkCudaErrors(hipStreamSynchronize(stream));
	checkCudaErrors(hipEventRecord(start, stream));

	int nIter = 100;	// How many times to run kernel

	// Launch kernel (repeat nIter times so we can obtain average run time)
	for (int i = 0; i < nIter; i++) {
		Histogram_Computation << < dim_grid, dim_block >> > (device_input, device_bins, input_size, num_bins);
		checkCudaErrors(hipDeviceSynchronize());
	}

	printf("\nGPU Histogram Computation Complete\n");

	// Record the stop event
	checkCudaErrors(hipEventRecord(stop, stream));

	// Wait for the stop event to complete
	checkCudaErrors(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

	// Compute and print the performance
	float msecPerHistogram = msecTotal / nIter;
	printf("\nTime = %.3f msec", msecPerHistogram);

	// Copy matrix values from device to host
	checkCudaErrors(hipMemcpy(host_bins, device_bins, bin_bytes, hipMemcpyDeviceToHost));

	for (int i = 0; i < num_bins; i++) {
		printf("\nBin Output = %d", host_bins[i]);
	}

	int tmp = 0;
	for (int i = 0; i < num_bins; i++) {
		tmp += host_bins[i];
	}

	printf("\nOutput = %d", tmp);

	// Free memory in device
	checkCudaErrors(hipFree(device_input));
	checkCudaErrors(hipFree(device_bins));

	// Free memory in host
	free(host_input);
	free(host_bins);
}
