#include "hip/hip_runtime.h"
// CUDA Histogram Computation
// Tim Demetriades
// CPE 810 - GPU & Multicore Programming
// Professor Feng
// Stevens Institute of Technology

#include <hip/hip_runtime.h>
#include <>
#include "hip/hip_runtime_api.h"

//for __syncthreads() and atomicadd
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#include <stdio.h>
#include <cstdlib>
#include <time.h>	// for CPU timer
#include <math.h>	// for power function

// Statically allocate shared memory
#define SHARED_MEM_SIZE 16	// Size of each block

// Kernel function for Histogram Computation using interleaved portioning (memory coalescing)
__global__ void Histogram_GPU_1(unsigned int* device_input, unsigned int* device_bins, unsigned long input_size, unsigned int bin_size) {

	// Get thread id
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for (unsigned int i = tid; i < input_size; i += (blockDim.x * gridDim.x)) {	// blockDim.x * gridDim.x = total number of threads for each kernel invocation
		if (device_input[i] >= 0 && device_input[i] < 1024) {	// Boundary condition
			atomicAdd(&device_bins[device_input[i] / bin_size], 1);
		}
	}
}

// Kernel function for Histogram Computation using shared memory (privatization)
__global__ void Histogram_GPU_2(unsigned int* device_input, unsigned int* device_bins, unsigned long input_size, unsigned int bin_size, unsigned int num_bins) {

	// Get thread id
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Private bins
	__device__ __shared__ int device_bins_private[SHARED_MEM_SIZE];
	// Initialize private bins to 0
	for (unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
		device_bins_private[binIdx] = 0;
	}
	__syncthreads();

	// Compute histogram
	for (unsigned int i = tid; i < input_size; i += blockDim.x * gridDim.x) {
		if (device_input[i] >= 0 && device_input[i] < 1024) {	// Boundary condition
			atomicAdd(&device_bins_private[device_input[i] / bin_size], 1);
		}
	}
	__syncthreads();

	// Move from shared to global memory
	for (unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
		atomicAdd(&device_bins[binIdx], device_bins_private[binIdx]);
	}
}

// Kernel function for Histogram Computation using shared memory (privatization) and aggregation
__global__ void Histogram_GPU_3(unsigned int* device_input, unsigned int* device_bins, unsigned long input_size, unsigned int bin_size, unsigned int num_bins) {

	// Get thread id
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Private bins
	__device__ __shared__ int device_bins_private[SHARED_MEM_SIZE];
	// Initialize private bins to 0
	for (unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
		device_bins_private[binIdx] = 0;
	}
	__syncthreads();

	unsigned int prev_index = -1;		// Tracks index of histogram element whose updates have been aggregated (-1 so it won't have chance of matching value in bin)
	unsigned int accumulator = 0;		// Keeps track of number of updates aggregated so far (0 means no updates have been aggregated)

	// Compute histogram
	for (unsigned int i = tid; i < input_size; i += blockDim.x * gridDim.x) {
		if (device_input[i] >= 0 && device_input[i] < 1024) {	// Boundary condition
			unsigned int current_index = device_input[i] / bin_size;		
			if (current_index != prev_index) {		// Compare index of histogram element to be updated with index of one currently being aggregated
				if (accumulator >= 0) {
					atomicAdd(&device_bins_private[device_input[i] / bin_size], accumulator);		// Current and previous are different, so add accumulator to value in bin
					accumulator = 1;
					prev_index = current_index;
				}
			}
			else {
				accumulator ++;		// Current and previous match so increment accumulator
			}
		}
	}
	__syncthreads();

	// Move from shared to global memory
	for (unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
		atomicAdd(&device_bins[binIdx], device_bins_private[binIdx]);
	}
}

// Histogram Computation on CPU
void Histogram_CPU(unsigned int* host_input, unsigned int input_size, unsigned int bin_size, unsigned int * host_bins) {
	for (int i = 0; i < input_size; i++) {
		host_bins[host_input[i] / bin_size]++;
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char* argv[]) {
	unsigned int num_bins = 0;
	unsigned long input_size = 0;

	if (argc == 4) {		// 4 arguments expected (filename, -i, <BinNum>, <VecDim>)
		int bin_exponent = atoi(argv[2]);
		if (bin_exponent < 2 || bin_exponent > 8) {
			printf("\nPlease make sure <BinNum> is between 2 and 8.\n");
			exit(EXIT_FAILURE);
		}
		// Set number of bins 
		num_bins = pow(2, bin_exponent);
		printf("\nThere will be %d bins \n", num_bins);

		// Set number of input elements
		input_size = atoi(argv[3]);
		printf("Input vector is %d elements long \n", input_size);
	}
	else if (argc > 4) {
		printf("\nToo many arguments provided. \n");
		printf("Enter arguments like this: \n");
		printf("-i <BinNum> <VecDim> \n");
		exit(EXIT_FAILURE);
	}
	else {
		printf("\n3 arguments expected. \n");
		printf("Enter arguments like this: \n");
		printf("-i <BinNum> <VecDim> \n");
		exit(EXIT_FAILURE);
	}

	// Set number of elements per bin
	unsigned int bin_size = 1024 / num_bins;		// 1024 is the max possible input element
	printf("Each bin will contain %u elements\n", bin_size);

	// Size in bytes of input vector
	size_t input_bytes = input_size * sizeof(int);

	// Size in bytes of bins
	size_t bin_bytes = num_bins * sizeof(int);

	// Allocate host memory for input vector and bins
	unsigned int* host_input;
	unsigned int* host_bins;
	unsigned int* host_bins_cpu;
	host_input = (unsigned int*)malloc(input_bytes);
	host_bins = (unsigned int*)malloc(bin_bytes);
	host_bins_cpu = (unsigned int*)malloc(bin_bytes);

	// Allocate device memory for input vector and bins
	unsigned int* device_input;
	unsigned int* device_bins;
	checkCudaErrors(hipMalloc((void**)&device_input, input_bytes));
	checkCudaErrors(hipMalloc((void**)&device_bins, bin_bytes));

	// Initialize input vector with ints between 0~1024
	srand((unsigned int)time(NULL));		// Assigns seed to make random numbers change
	for (int i = 0; i < input_size; i++) {
		host_input[i] = rand() % 1024;
	}

	// Initialize bins with 0s
	for (int i = 0; i < num_bins; i++) {
		host_bins[i] = 0;
	}
	for (int i = 0; i < num_bins; i++) {
		host_bins_cpu[i] = 0;
	}

	// Print input vector
	/*printf("\nInput vector:\n");
	for (int i = 0; i < input_size; i++) {
		printf("%d\t", host_input[i]);
	}*/

	// Copy matrix values from host to device
	checkCudaErrors(hipMemcpy(device_input, host_input, input_bytes, hipMemcpyHostToDevice));		// dest, source, size in bytes, direction of transfer

	// Set Grid and Block sizes
	int block_size = 16;		// Threads per block
	int grid_size = input_size / block_size;
	dim3 dim_block(block_size);
	dim3 dim_grid(grid_size);

	// Record the start event (for timing GPU calculations)
	hipStream_t stream;
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	checkCudaErrors(hipStreamSynchronize(stream));
	checkCudaErrors(hipEventRecord(start, stream));

	int nIter = 1;	// How many times to run kernel

	printf("\nStarting Histogram Computation on GPU\n");

	// Launch kernel (repeat nIter times so we can obtain average run time)
	for (int i = 0; i < nIter; i++) {
		//Histogram_GPU_1<<<dim_grid, dim_block>>>(device_input, device_bins, input_size, bin_size);
		Histogram_GPU_2<<<dim_grid, dim_block>>>(device_input, device_bins, input_size, bin_size, num_bins);
		//Histogram_GPU_3<<<dim_grid, dim_block>>>(device_input, device_bins, input_size, bin_size, num_bins);
	}

	printf("\n\GPU Histogram Computation Complete\n");

	// Record the stop event
	checkCudaErrors(hipEventRecord(stop, stream));

	// Wait for the stop event to complete
	checkCudaErrors(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

	// Compute and print the performance
	float msecPerHistogram = msecTotal / nIter;
	printf("\nGPU Histogram Computation took %.3f msec\n", msecPerHistogram);

	// Copy matrix values from device to host
	checkCudaErrors(hipMemcpy(host_bins, device_bins, bin_bytes, hipMemcpyDeviceToHost));

	//// Print GPU results
	//printf("\nGPU Results: \n");
	//for (int i = 0; i < num_bins; i++) {
	//	printf("\nBins %d = %u", i, host_bins[i]);
	//}

	//int sum_bins = 0;
	//for (int i = 0; i < num_bins; i++) {
	//	sum_bins += host_bins[i];
	//}
	//printf("\n\nSummation of all the bins = %d\n", sum_bins);

	//Start CPU timer
	double time_taken_cpu = 0.0;
	clock_t begin_cpu = clock();

	// Calculate histogram on CPU
	printf("\nStarting Histogram Computation on CPU\n");
	Histogram_CPU(host_input, input_size, bin_size, host_bins_cpu);
	printf("\nCPU Histogram Computation Complete\n");

	clock_t end_cpu = clock();
	time_taken_cpu += (double)(end_cpu - begin_cpu) / CLOCKS_PER_SEC * 1000;	// in milliseconds
	printf("\nCPU Histogram Computation took %.3f msec\n", time_taken_cpu);

	//// Print CPU results
	//printf("\nCPU Results: \n");
	//for (int i = 0; i < num_bins; i++) {
	//	printf("\nBins %d = %u", i, host_bins_cpu[i]);
	//}

	//int sum_bins_cpu = 0;
	//for (int i = 0; i < num_bins; i++) {
	//	sum_bins_cpu += host_bins_cpu[i];
	//}
	//printf("\n\nSummation of all the bins = %d\n", sum_bins_cpu);

	// Check if GPU and CPU histograms match
	bool check = 0;
	for (int i = 0; i < num_bins; i++) {			// For every value in the arrays
		if (host_bins[i] != host_bins_cpu[i]) {		// Check if they match and if not set a flag
			check = 1;
		}
	}

	if (check == 1) {
		printf("\nGPU and CPU histograms do not match!\n");
	}
	else {
		printf("\nGPU and CPU histograms match!\n");
	}

	// Free memory in device
	hipFree(device_input);
	hipFree(device_bins);

	// Free memory in host
	free(host_input);
	free(host_bins);
}
